/*! @file fdtd.cu

  @brief This is the entry point of the file.
  */
#include "fdtd.h"
#include "hip/hip_runtime.h"
#include "cpu_anim.h"
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include "h5save.h"
#include <stdio.h>
#include <pthread.h>
#include "datablock.h"
#include "kernels.cuh"
#include "constants.h"
#include <thrust/fill.h>
#include <algorithm>
#include "tm_mode.h"
#include "pml_mode.h"
#include "drude_mode.h"
#include <fstream>
#include <assert.h>
#include <string>
#include "common_functions.h"

using namespace std;
pthread_mutex_t mutexcopy;

/** @brief Calls the gpu kernels in order.
  * Different types of simulation.
  */
void anim_gpu(Datablock *d, int ticks){
    if(d->simulationType == TM_SIMULATION)
        anim_gpu_tm(d, ticks);
    else if(d->simulationType == TM_PML_SIMULATION)
        anim_gpu_pml_tm(d, ticks);
    else if(d->simulationType == DRUDE_SIMULATION)
        anim_gpu_drude(d, ticks);
}

/*! @brief Clears memory when the simulation is done. */
void anim_exit(Datablock *d){
    if(d->simulationType == TM_SIMULATION)
        clear_memory_TM_simulation(d);
    else if(d->simulationType == TM_PML_SIMULATION)
        clear_memory_TM_PML_simulation(d);
    else if(d->simulationType == DRUDE_SIMULATION)
        clear_memory_drude_simulation(d);

}

/*! @brief Allocates memory for the simulation depending on the type
   of the simulation.
*/
size_t allocate_memory(Datablock *data, Structure structure){
    if(data->simulationType == TM_SIMULATION)
        return allocateTMMemory(data, structure);
    else if(data->simulationType == TM_PML_SIMULATION)
        return tm_pml_allocate_memory(data, structure);
    else if(data->simulationType == DRUDE_SIMULATION)
        return allocate_drude_memory(data, structure);
    return 0;
}

/*! @brief Initializes the memory for simulation.*/
void initializeArrays(Datablock *data, Structure structure, ifstream &fs){
    if(data->simulationType == TM_SIMULATION)
        initialize_TM_arrays(data, structure, fs);
    else if(data->simulationType == TM_PML_SIMULATION)
        tm_pml_initialize_arrays(data, structure, fs);
    else if(data->simulationType == DRUDE_SIMULATION)
        initialize_drude_arrays(data, structure);
}

/*! @brief Clears all the constants initially declared.
  This method is called once all the coefficients are
  calculated.
*/
void clear_memory_constants(Datablock *data){
    if(data->simulationType == TM_SIMULATION)
        tm_clear_memory_constants(data);
    else if(data->simulationType == TM_PML_SIMULATION)
        tm_pml_clear_memory_constants(data);
    else if(data->simulationType == DRUDE_SIMULATION)
        drude_clear_memory_constants(data);
}

/*!
  @brief Calculates the coefficients for each simulation.
*/
void calculate_coefficients(Datablock *data, Structure structure){
    dim3 blocks((structure.x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (structure.y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);
    if(data->simulationType == TM_SIMULATION){
        tm_getcoeff<<<blocks, threads>>>(data->constants[MUINDEX],
                                         data->constants[EPSINDEX],
                                         data->constants[SIGMAINDEX],
                                         data->constants[SIGMA_STAR_INDEX],
                                         data->coefs[0],
                                         data->coefs[1],
                                         data->coefs[2],
                                         data->coefs[3]
                                         );
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
    }

    else if(data->simulationType == TM_PML_SIMULATION)
    {
        pml_tm_get_coefs<<<blocks, threads>>>(data->constants[MUINDEX],
                                              data->constants[EPSINDEX],
                                              data->constants[SIGMAINDEX_X],
                                              data->constants[SIGMAINDEX_Y],
                                              data->constants[SIGMA_STAR_INDEX_X],
                                              data->constants[SIGMA_STAR_INDEX_Y],
                                              data->coefs[0],
                                              data->coefs[1],
                                              data->coefs[2],
                                              data->coefs[3],
                                              data->coefs[4],
                                              data->coefs[5],
                                              data->coefs[6],
                                              data->coefs[7]);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
    }

    else if(data->simulationType == DRUDE_SIMULATION){
        drude_get_coefs<<<blocks, threads>>>(data->constants[MUINDEX],
                                         data->constants[EPSINDEX],
                                         data->constants[SIGMAINDEX],
                                         data->constants[SIGMA_STAR_INDEX],
                                         data->constants[GAMMA_INDEX],
                                         data->constants[OMEGAP_INDEX],
                                         data->coefs[0],
                                         data->coefs[1],
                                         data->coefs[2],
                                         data->coefs[3],
                                         data->coefs[4],
                                         data->coefs[5],
                                         data->coefs[6]);
        hipDeviceSynchronize();
        checkCudaErrors(hipGetLastError());
    }
}

/*! @brief entry point */
int main(int argc, char **argv){
    assert(argc == 2);
    ifstream fs;
    FILE *fp = fopen("logFile.txt", "w");
    assert(fp != NULL);
    fs.open(argv[1]);
    assert(fs.is_open());
    
    string simulation_name;
    fs>>simulation_name;

    int simulation_type;
    fs>>simulation_type;

    int output_type;
    fs>>output_type;

    Datablock data(simulation_type, output_type);
    data.simulation_name = simulation_name;

    float dx;
    fs>>dx;

    int hdf5_count = 0;
    if(output_type == 1){
        fs>>hdf5_count;
    }

    float courant = 0.5;
    float dt =  courant * dx / LIGHTSPEED;
    fprintf(fp, "In fdtd.cu ...\n");
    fprintf(fp, "dt = %E\n", dt);

    int xdim, ydim;
    fs>>xdim>>ydim;
    Structure structure(xdim, ydim, dx, dt);
    fprintf(fp, "The grid size is %ld.\n\n", (long) (xdim * ydim));

    CPUAnimBitmap bitmap(structure.x_index_dim, structure.x_index_dim,
                            &data); /* bitmap structure */
    data.bitmap = &bitmap;
    data.totalTime = 0;
    data.frames = 0;
    checkCudaErrors(hipEventCreate(&data.start, 1) );
    checkCudaErrors(hipEventCreate(&data.stop, 1) );

    size_t pitch;
    // memory allocation for fields, coefs, and consts. 
    fprintf(fp, "Allocating memory ...\n");
    pitch = allocate_memory(&data, structure);
    fprintf(fp, "Done.\n\n");

    structure.pitch = pitch;
    
    copy_symbols(&structure);
    fprintf(fp, "pitch = %d\n\n", (int)pitch);
    data.structure = &structure;
    
    // initialising arrays
    fprintf(fp, "Initializing Arrays ...\n");
    initializeArrays(&data, structure, fs);
    fprintf(fp, "Done.\n\n");

    // get the coefficients
    fprintf(fp, "Calculating coefficients ...\n");
    calculate_coefficients(&data, structure);
    clear_memory_constants(&data);
    fprintf(fp, "Done.\n\n");

    // set the sources
    HostSources host_sources;
    DeviceSources device_sources;
    long long x, y, source_type;
    float mean, variance;

    fprintf(fp, "Setting the sources on the host ...\n");
    while(!fs.eof()){
        fs >> x >> y >> source_type >> mean >> variance;
        host_sources.add_source(x, y, source_type, mean, variance);
    }
    fprintf(fp, "Done.\n\n");

    data.sources = &device_sources;
    fprintf(fp, "Copying the sources on the device ...\n");
    copy_sources_device_to_host(&host_sources, &device_sources);
    fprintf(fp, "Done.\n\n");
    
    if(data.outputType == OUTPUT_HDF5){
        pthread_mutex_init(&mutexcopy, NULL);
        for(long i=0; i < hdf5_count; i++){
            anim_gpu(&data, 0);
        }
    }

    if(data.outputType == OUTPUT_ANIM){
        bitmap.anim_and_exit((void (*)(void *, int)) anim_gpu,
                            (void (*)(void *)) anim_exit);
    }
}
