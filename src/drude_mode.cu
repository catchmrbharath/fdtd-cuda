/*! @file drude_mode.cu
   @author Bharath M R

   @brief Contains the functions for the drude mode.
*/
#include "drude_mode.h"
#include "constants.h"
#include "h5save.h"
#include "fdtd.h"
#include <pthread.h>
/**
  Main entry point for the fdtd calculations

  @param d The datablock structure
  @param ticks Represents the number of times the function has run.
*/
void anim_gpu_drude(Datablock *d, int ticks){
    assert(d != NULL);
    float err = hipEventRecord(d->start, 0);

    dim3 blocks((d->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (d->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    dim3 source_threads(64, 1);
    dim3 source_blocks((d->sources->size + 63) / 64, 1);
    CPUAnimBitmap *bitmap = d->bitmap;
    static long time_ticks = 0;

    for(int i=0;i<100;i++){
        time_ticks += 1;
        make_copy<<<blocks, threads>>>(d->fields[DRUDE_EZOLD],
                                       d->fields[DRUDE_EZFIELD]);

        update_drude_ez<<<blocks, threads>>>(d->fields[DRUDE_EZFIELD],
                                             d->fields[DRUDE_HXFIELD],
                                             d->fields[DRUDE_HYFIELD],
                                             d->fields[DRUDE_JFIELD],
                                             d->coefs[2],
                                             d->coefs[3],
                                             d->coefs[4]);

        copy_sources<<<source_blocks, source_threads>>>(
                d->fields[DRUDE_EZFIELD],
                d->sources->x_source_position,
                d->sources->y_source_position,
                d->sources->source_type,
                d->sources->mean,
                d->sources->variance,
                d->sources->size,
                time_ticks);

        update_drude_jz<<<blocks, threads>>>(d->fields[DRUDE_JFIELD],
                                             d->fields[DRUDE_EZFIELD],
                                             d->fields[DRUDE_EZOLD],
                                             d->coefs[5],
                                             d->coefs[6]);

        update_Hx<<<blocks, threads>>>(d->fields[DRUDE_HXFIELD],
                                        d->fields[DRUDE_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);

        update_Hy<<<blocks, threads>>>(d->fields[DRUDE_HYFIELD],
                                        d->fields[DRUDE_EZFIELD],
                                        d->coefs[0],
                                        d->coefs[1]);
    }
    
    if(d->outputType == OUTPUT_ANIM)
    {
        float_to_color<<<blocks, threads>>> (d->output_bitmap,
                d->fields[DRUDE_EZFIELD]);
        checkCudaErrors(hipMemcpy2D(bitmap->get_ptr(),
                                    sizeof(float) * d->structure->x_index_dim,
                                    d->output_bitmap,
                                    d->structure->pitch,
                                    sizeof(float) * d->structure->x_index_dim,
                                    d->structure->y_index_dim,
                                    hipMemcpyDeviceToHost));
    }

    if(d->outputType == OUTPUT_HDF5)
    {
        pthread_t thread;
        /*Copy back to cpu memory */
        /*Create a lock */
        pthread_mutex_lock(&mutexcopy);
        checkCudaErrors(hipMemcpy2D(d->save_field,
                                   sizeof(float) * d->structure->x_index_dim,
                                   d->fields[DRUDE_EZFIELD],
                                   d->structure->pitch,
                                   sizeof(float) * d->structure->x_index_dim,
                                   d->structure->y_index_dim,
                                   hipMemcpyDeviceToHost));
        pthread_mutex_unlock(&mutexcopy);
        pthread_create(&thread, NULL, &create_new_dataset, (void *)d);
        create_new_dataset(d);
    }

    d->present_ticks = time_ticks;
    checkCudaErrors(hipEventRecord(d->stop, 0) );
    checkCudaErrors(hipEventSynchronize(d->stop));
    float elapsedTime = 1;
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, d->start, d->stop));
    d->totalTime +=elapsedTime;
    d->frames +=1;
    printf("Average time per frame: %3.1f ms\n", elapsedTime);
}

/*! @brief Clears the constants after the coefficients are calculated
  */
void drude_clear_memory_constants(Datablock *d){
    hipFree(d->constants[SIGMAINDEX]);
    hipFree(d->constants[SIGMA_STAR_INDEX]);
    hipFree(d->constants[EPSINDEX]);
    hipFree(d->constants[MUINDEX]);
    hipFree(d->constants[GAMMA_INDEX]);
    hipFree(d->constants[OMEGAP_INDEX]);
}

/*! @ brief Clears the memory after the simulation. */

//TODO: Replace the calls with a for loop.
void clear_memory_drude_simulation(Datablock *d){
    hipFree(d->fields[DRUDE_EZFIELD]);
    hipFree(d->fields[DRUDE_HXFIELD]);
    hipFree(d->fields[DRUDE_HYFIELD]);
    hipFree(d->fields[DRUDE_JFIELD]);
    hipFree(d->fields[DRUDE_EZOLD]);
    hipFree(d->coefs[0]);
    hipFree(d->coefs[1]);
    hipFree(d->coefs[2]);
    hipFree(d->coefs[3]);
    hipFree(d->coefs[4]);
    hipFree(d->coefs[5]);
    hipFree(d->coefs[6]);
    hipFree(d->sources->x_source_position);
    hipFree(d->sources->y_source_position);
    hipFree(d->sources->source_type);
    hipFree(d->sources->mean);
    hipFree(d->sources->variance);
    checkCudaErrors(hipEventDestroy(d->start) );
    checkCudaErrors(hipEventDestroy(d->stop) );
}

/*! @brief Allocates the memory for the simulation */
size_t allocate_drude_memory(Datablock *data, Structure structure){
    printf("The size of the structure is %ld \n", structure.size());
    printf("Allocation Memory\n");
    size_t pitch; //! pitch is the row size in bytes.

    checkCudaErrors(hipMallocPitch( (void **) &data->output_bitmap,
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));

    for(int i = 0;i < 5;i++){
        checkCudaErrors(hipMallocPitch( (void **) &data->fields[i],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    }

    for(int i = 0; i < 6; i++){
        checkCudaErrors(hipMallocPitch( (void **) &data->constants[i],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    }

    for(int i = 0;i < 7; i++){
        checkCudaErrors(hipMallocPitch( (void **) &data->coefs[i],
                    &pitch, sizeof(float) * structure.x_index_dim,
                    sizeof(float) * structure.y_index_dim ));
    }
    return pitch;
}

/*! @brief Allocates the memory for the simulation */
void initialize_drude_arrays(Datablock *data, Structure structure){
    long size = structure.grid_size();
    printf("%ld\n", size);
    printf("%d\n", structure.x_index_dim);
    printf("%d\n", structure.y_index_dim);
    printf("Initializing arrays\n");

    // FIXME: Temporary fix for populating values.
    float * temp = (float *)malloc(sizeof(float) * size);
    std::fill_n(temp, size, MU);
    checkCudaErrors(hipMemcpy2D(data->constants[MUINDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) * structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, EPSILON);
    checkCudaErrors(hipMemcpy2D(data->constants[EPSINDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) * structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, 0.0);
    checkCudaErrors(hipMemcpy2D(data->constants[SIGMAINDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) * structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, 0.0);
    checkCudaErrors(hipMemcpy2D(data->constants[SIGMA_STAR_INDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) *  structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));


    std::fill_n(temp, size, 2.0 * PI * 2e15);
    checkCudaErrors(hipMemcpy2D(data->constants[OMEGAP_INDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) *  structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));

    std::fill_n(temp, size, 57e12);
    checkCudaErrors(hipMemcpy2D(data->constants[GAMMA_INDEX], structure.pitch,
                temp, sizeof(float) * structure.x_index_dim,
                sizeof(float) *  structure.x_index_dim,
                structure.y_index_dim,
                hipMemcpyHostToDevice));
   dim3 blocks((data->structure->x_index_dim + BLOCKSIZE_X - 1) / BLOCKSIZE_X,
                (data->structure->y_index_dim + BLOCKSIZE_Y - 1) / BLOCKSIZE_Y);
    dim3 threads(BLOCKSIZE_X, BLOCKSIZE_Y);

    initialize_array<<<blocks, threads>>>(data->fields[DRUDE_EZFIELD], 0);
    initialize_array<<<blocks, threads>>>(data->fields[DRUDE_EZOLD], 0);
    initialize_array<<<blocks, threads>>>(data->fields[DRUDE_HXFIELD], 0);
    initialize_array<<<blocks, threads>>>(data->fields[DRUDE_HYFIELD], 0);
    initialize_array<<<blocks, threads>>>(data->fields[DRUDE_JFIELD], 0);
}
